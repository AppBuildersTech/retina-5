#include "KernelInvoker.cuh"

extern int*   h_no_sensors;
extern int*   h_no_hits;
extern int*   h_sensor_Zs;
extern int*   h_sensor_hitStarts;
extern int*   h_sensor_hitNums;
extern unsigned int* h_hit_IDs;
extern float* h_hit_Xs;
extern float* h_hit_Ys;
extern float* h_hit_Zs;

hipError_t invokeParallelSearch(
    dim3                         numBlocks,
    dim3                         numThreads,
    const std::vector<uint8_t> & input,
    std::vector<uint8_t>       & solution) {
  // For now, just perform what we did before
  // (backwards compatibility)
  int* h_track_indexes;
  int* num_tracks;
  Track* tracks;

  DEBUG << "Input pointer: " 
    << std::hex << "0x" << (long long int) &(input[0])
    << std::dec << std::endl;

  setHPointersFromInput(const_cast<uint8_t*>(&input[0]), input.size());
  printInfo();

  std::map<int, int> zhit_to_module;
  if (logger::ll.verbosityLevel > 0){
    // map to convert from z of hit to module
    for(int i=0; i<*h_no_sensors; ++i){
      const int z = h_sensor_Zs[i];
      zhit_to_module[z] = i;
    }

    // Some hits z may not correspond to a sensor's,
    // but be close enough
    for(int i=0; i<*h_no_hits; ++i){
      const int z = h_hit_Zs[i];
      if (zhit_to_module.find(z) == zhit_to_module.end()){
        const int sensor = findClosestModule(z, zhit_to_module);
        zhit_to_module[z] = sensor;
      }
    }
  }

  // int* h_prevs, *h_nexts;
  // Histo histo;

  char*  dev_input             = 0;
  int*   dev_num_tracks        = 0;
  int*   dev_track_indexes     = 0;
  Track* dev_tracks            = 0;
  bool*  dev_track_holders     = 0;
  int*   dev_prevs             = 0;
  int*   dev_nexts             = 0;
  int*   dev_tracks_to_process = 0;
  hipError_t cudaStatus = hipSuccess;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaCheck( hipSetDevice(0) );

  // Allocate memory
  // Allocate CPU buffers
  tracks = (Track*) malloc(MAX_TRACKS * sizeof(Track));
  //solution.resize(MAX_TRACKS * sizeof(Track));
  //tracks = (Track*) &(solution[0]);
  num_tracks = (int*) malloc(sizeof(int));

  int* h_prevs = (int*) malloc(h_no_hits[0] * sizeof(int));
  int* h_nexts = (int*) malloc(h_no_hits[0] * sizeof(int));
  bool* h_track_holders = (bool*) malloc(MAX_TRACKS * sizeof(bool));
  h_track_indexes = (int*) malloc(MAX_TRACKS * sizeof(int));

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks, MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_track_holders, MAX_TRACKS * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_track_indexes, MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_process, MAX_TRACKS * sizeof(int)));

  cudaCheck(hipMalloc((void**)&dev_prevs, h_no_hits[0] * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_nexts, h_no_hits[0] * sizeof(int)));

  // Copy input file from host memory to GPU buffers
  cudaCheck(hipMalloc((void**)&dev_input, input.size()));
  cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));

  // memcpys
  cudaCheck(hipMemcpy(dev_input, &(input[0]), input.size(), hipMemcpyHostToDevice));

  // Launch a kernel on the GPU with one thread for each element.
  prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts, dev_track_holders);

  // gpuKalman
  DEBUG << "gpuKalman" << std::endl;
  hipEvent_t start_kalman, start_postprocess, stop;
  float t0, t1, t2;

  hipEventCreate(&start_kalman);
  hipEventCreate(&start_postprocess);
  hipEventCreate(&stop);

  hipEventRecord(start_kalman, 0 );

  gpuKalman<<<numBlocks, numThreads>>>(dev_tracks, dev_track_holders);

  hipEventRecord(start_postprocess);


  DEBUG << "postProcess" << std::endl;
  postProcess<<<1, numThreads>>>(dev_tracks, dev_track_holders, dev_track_indexes, dev_num_tracks, dev_tracks_to_process);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  hipEventElapsedTime( &t0, start_kalman, start_postprocess );
  hipEventElapsedTime( &t1, start_postprocess, stop );
  hipEventElapsedTime( &t2, start_kalman, stop );
  hipEventDestroy( start_kalman );
  hipEventDestroy( start_postprocess );
  hipEventDestroy( stop );

  // Get results
  cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(h_track_indexes, dev_track_indexes, MAX_TRACKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));

  // number of tracks after stage#1
  int no_tracks_stage1 = 0;
  for(int i=0; i<h_no_hits[0]; ++i)
    if(h_track_holders[i])
      ++no_tracks_stage1;

  // copy selected track to the solution vector
  if (*num_tracks > 0) {
    solution.resize(*num_tracks * sizeof(Track));
    Track * solutionTracks = (Track*)&solution[0];
    for (size_t i = 0; i != *num_tracks; ++i)
      solutionTracks[i] = tracks[h_track_indexes[i]];
  }

  // print debug info
  if (logger::ll.verbosityLevel > 0){
    for(int i=0; i<num_tracks[0]; ++i){
      printTrack(tracks, h_track_indexes[i], i, zhit_to_module);
    }
  }

  DEBUG << "Processed " << num_tracks[0] << " tracks" << std::endl;

  free(h_prevs);
  free(h_nexts);
  free(h_track_holders);
  free(tracks);
  free(num_tracks);

  return cudaStatus;
}

/**
 * Prints tracks
 * Track #n, length <length>:
 *  <ID> module <module>, x <x>, y <y>, z <z>
 * 
 * @param tracks      
 * @param trackNumber 
 */
void printTrack(Track* tracks, const int trackID,
  const int trackNumber, const std::map<int, int>& zhit_to_module){

  const Track t = tracks[trackID];
  DEBUG << "Track #" << trackNumber << ", length " << (int) t.hitsNum << std::endl;

  for(int i=0; i<t.hitsNum; ++i){
    const int hitNumber = t.hits[i];
    const unsigned int id = h_hit_IDs[hitNumber];
    const float x = h_hit_Xs[hitNumber];
    const float y = h_hit_Ys[hitNumber];
    const float z = h_hit_Zs[hitNumber];
    const int module = zhit_to_module.at((int) z);

    DEBUG << " " << std::setw(8) << id
      << " module " << std::setw(2) << module
      << ", x " << std::setw(6) << x
      << ", y " << std::setw(6) << y
      << ", z " << std::setw(6) << z << std::endl;
  }

  DEBUG << std::endl;
}

/**
 * The z of the hit may not correspond to any z in the sensors.
 * @param  z              
 * @param  zhit_to_module 
 * @return                sensor number
 */
int findClosestModule(const int z, const std::map<int, int>& zhit_to_module){
  if (zhit_to_module.find(z) != zhit_to_module.end())
    return zhit_to_module.at(z);

  int error = 0;
  while(true){
    error++;
    const int lowerAttempt = z - error;
    const int higherAttempt = z + error;

    if (zhit_to_module.find(lowerAttempt) != zhit_to_module.end()){
      return zhit_to_module.at(lowerAttempt);
    }
    if (zhit_to_module.find(higherAttempt) != zhit_to_module.end()){
      return zhit_to_module.at(higherAttempt);
    }
  }
}

void printOutAllSensorHits(int* prevs, int* nexts){
  DEBUG << "All valid sensor hits: " << std::endl;
  for(int i=0; i<h_no_sensors[0]; ++i){
    for(int j=0; j<h_sensor_hitNums[i]; ++j){
      int hit = h_sensor_hitStarts[i] + j;

      if(nexts[hit] != -1){
        DEBUG << hit << ", " << nexts[hit] << std::endl;
      }
    }
  }
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts){
  for(int i=0; i<h_sensor_hitNums[sensorNumber]; ++i){
    int hstart = h_sensor_hitStarts[sensorNumber];

    DEBUG << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
  }
}

void printInfo() {
  DEBUG << "Read info:" << std::endl
    << " no sensors: " << h_no_sensors[0] << std::endl
    << " no hits: " << h_no_hits[0] << std::endl
    << "First 5 sensors: " << std::endl;

  for (int i=0; i<5; ++i){
    DEBUG << " Zs: " << h_sensor_Zs[i] << std::endl
      << " hitStarts: " << h_sensor_hitStarts[i] << std::endl
      << " hitNums: " << h_sensor_hitNums[i] << std::endl << std::endl;
  }

  DEBUG << "First 5 hits: " << std::endl;

  for (int i=0; i<5; ++i){
    DEBUG << " hit_id: " << h_hit_IDs[i] << std::endl
      << " hit_X: " << h_hit_Xs[i] << std::endl
      << " hit_Y: " << h_hit_Ys[i] << std::endl
      << " hit_Z: " << h_hit_Zs[i] << std::endl << std::endl;
  }
}

void getMaxNumberOfHits(char*& input, int& maxHits){
  int* l_no_sensors = (int*) &input[0];
  int* l_no_hits = (int*) (l_no_sensors + 1);
  int* l_sensor_Zs = (int*) (l_no_hits + 1);
  int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
  int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

  maxHits = 0;
  for(int i=0; i<l_no_sensors[0]; ++i){
    if(l_sensor_hitNums[i] > maxHits)
      maxHits = l_sensor_hitNums[i];
  }
}
