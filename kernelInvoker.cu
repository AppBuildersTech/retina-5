
#include "kernelInvoker.cuh"
#include "kernel.cuh"

#define cudaCheck(stmt) do {										\
        hipError_t err = stmt;										\
        if (err != hipSuccess) {									\
            std::cerr << "Failed to run " << #stmt << std::endl;    \
            return err;										        \
        }															\
    } while(0)

// Helper function for using CUDA to add vectors in parallel.
hipError_t invokeParallelSearch(int numBlocks, int numThreads,
	char* input, int size, Track*& tracks, int*& num_tracks){
    
	char *dev_input = 0;
	int* dev_num_tracks = 0;
	Track *dev_tracks = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaCheck(hipSetDevice(0));
    
	// Allocate memory
	// Allocate CPU buffers
	tracks = (Track*) malloc(max_tracks * sizeof(Track));
	num_tracks = (int*) malloc(sizeof(int));

    // Allocate GPU buffers
    cudaCheck(hipMalloc((void**)&dev_tracks, max_tracks * sizeof(Track)));
    
    // Copy input file from host memory to GPU buffers
    cudaCheck(hipMalloc((void**)&dev_input, size));
    cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));
    
	// memcpys
    cudaCheck(hipMemcpy(dev_input, input, size, hipMemcpyHostToDevice));
    
    // Launch a kernel on the GPU with one thread for each element.
    parallelSearch<<<numBlocks, numThreads>>>(dev_tracks, dev_input, dev_num_tracks);
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaCheck(hipDeviceSynchronize());
    
	// cuda copy back
	cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	
    // Copy output vector from GPU buffer to host memory.
    cudaCheck(hipMemcpy(tracks, dev_tracks, num_tracks[0] * sizeof(Track), hipMemcpyDeviceToHost));
    
    return cudaStatus;
}
