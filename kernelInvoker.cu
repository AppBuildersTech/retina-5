
#include "kernelInvoker.cuh"
#include "kernel.cuh"

#define cudaCheck(stmt) do {										\
        hipError_t err = stmt;										\
        if (err != hipSuccess) {									\
            std::cerr << "Failed to run " << #stmt << std::endl;    \
            return err;										        \
        }															\
    } while(0)

// Helper function for using CUDA to add vectors in parallel.
hipError_t invokeParallelSearch(dim3 numBlocks, dim3 numThreads,
	char* input, int size, Track*& tracks, int*& num_tracks){
    
	char *dev_input = 0;
	int* dev_num_tracks = 0;
	Track *dev_tracks = 0;
	int* dev_prevs = 0;
	int* dev_nexts = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaCheck(hipSetDevice(0));
    
	// Allocate memory
	// Allocate CPU buffers
	tracks = (Track*) malloc(MAX_TRACKS * sizeof(Track));
	num_tracks = (int*) malloc(sizeof(int));

    // Allocate GPU buffers
    cudaCheck(hipMalloc((void**)&dev_tracks, MAX_TRACKS * sizeof(Track)));
	cudaCheck(hipMalloc((void**)&dev_prevs, h_no_hits[0] * sizeof(int)));
	cudaCheck(hipMalloc((void**)&dev_nexts, h_no_hits[0] * sizeof(int)));
    
    // Copy input file from host memory to GPU buffers
    cudaCheck(hipMalloc((void**)&dev_input, size));
    cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));
    
	// memcpys
    cudaCheck(hipMemcpy(dev_input, input, size, hipMemcpyHostToDevice));
    
    // Launch a kernel on the GPU with one thread for each element.
	prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts);
    neighboursFinder<<<numBlocks, numThreads>>>();
	neighboursCleaner<<<numBlocks, numThreads>>>();
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaCheck(hipDeviceSynchronize());
    
	// cuda copy back
	cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	
    // Copy output vector from GPU buffer to host memory.
    cudaCheck(hipMemcpy(tracks, dev_tracks, num_tracks[0] * sizeof(Track), hipMemcpyDeviceToHost));
    
    return cudaStatus;
}

void getMaxNumberOfHits(char*& input, int& maxHits){
	int* l_no_sensors = (int*) &input[0];
    int* l_no_hits = (int*) (l_no_sensors + 1);
    int* l_sensor_Zs = (int*) (l_no_hits + 1);
    int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
    int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

	maxHits = 0;
	for(int i=0; i<l_no_sensors[0]; ++i){
		if(l_sensor_hitNums[i] > maxHits)
			maxHits = l_sensor_hitNums[i];
	}
}