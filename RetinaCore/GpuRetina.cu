#include "hip/hip_runtime.h"
#include "GpuRetina.cuh"
#include <cstdio>
template<int BLOCK_SIZE>
__global__ void calculateRetina2d(
  const TrackProjection* tracks, 
  int tracksNum, 
  const double* hitsX,
  const double* hitsZ,
  int hitsNum, 
  double sharpness,
  double *values
)
{
  int trackId = blockIdx.x;
  unsigned int tid = threadIdx.x;
  const double trackX0 = tracks[trackId].x0;
  const double trackDx = tracks[trackId].dx;
  double sum = 0;
  for (int hitId = tid; hitId < hitsNum; hitId += BLOCK_SIZE)
  {
    const double hitX = hitsX[hitId];
    const double hitZ = hitsZ[hitId];
    const double shift = (hitX - trackX0 - hitZ * trackDx);
    sum += exp(-shift * shift * sharpness);
  }
  __shared__ double sdata[BLOCK_SIZE];
  for (unsigned int s = BLOCK_SIZE >> 1; s > 0; s >>= 1) 
  {
    if (tid < s) 
    {
      sdata[tid] += sdata[tid + s];
    }
  __syncthreads();
  }
  if (tid == 0)
  {
    values[trackId] = sdata[0];
  }
}

template<class T>
T* allocAndFetch(const T* data, int size)
{
  T* answer = nullptr;
  hipMalloc( (void**)&answer, sizeof(T) * size);
  hipMemcpy( answer, data, sizeof(T) * size, hipMemcpyHostToDevice);
  return answer;
}

void getRetinaDxGpu(
  const TrackProjection* tracks, 
  int tracksNum, 
  const double* hitsX,
  const double* hitsZ,
  int hitsNum, 
  double sharpness,
  double *values
) {
  TrackProjection* tracksGpu = allocAndFetch(tracks, tracksNum);
  double* hitsXGpu = allocAndFetch(hitsX, hitsNum);
  double* histZGpu = allocAndFetch(hitsZ, hitsNum);
  double* valuesGpu = nullptr;
  hipMalloc( (void**)&valuesGpu, sizeof(double) * tracksNum);
  calculateRetina2d<128><<<tracksNum, 128>>>(
    tracksGpu, 
    tracksNum, 
    hitsXGpu, 
    histZGpu, 
    hitsNum, 
    sharpness, 
    valuesGpu
  );
  hipMemcpy( valuesGpu, values, sizeof(double) * tracksNum, hipMemcpyDeviceToHost );
}


void getRetinaDxCpu(
  const TrackProjection* tracks, 
  int tracksNum, 
  const Hit* hits,
  int hitsNum, 
  double sharpness,
  double *values
) {
  int trackId, hitId;
  for (trackId = 0; trackId < tracksNum; ++trackId)
  {
    if (trackId % 1000 == 0)
      printf("%d\n", trackId);
    const double trackX0 = tracks[trackId].x0;
    const double trackDx = tracks[trackId].dx;
    double sum = 0;
    for (hitId = 0; hitId < hitsNum; hitId++)
    {
      const double hitX = hits[hitId].x;
      const double hitZ = hits[hitId].z;
      const double shift = (hitX - trackX0 - hitZ * trackDx);
      sum += exp(-shift * shift * sharpness);
    }
    values[trackId] = sum;
  }
}
/*
void getRetinaDy(
  const TrackProjection* tracks, 
  int tracksNum, 
  const Hit* hits,
  int hitsNum, 
  double sharpness,
  double *values
) {
  int trackId, hitId;
  for (trackId = 0; trackId < tracksNum; ++trackId)
  {
    if (trackId % 1000 == 0)
      printf("%d\n", trackId);
    const double trackX0 = track[hitId].x;
    const double trackDx = track[hitId].dx;
    double sum = 0;
    for (hitId = 0; hitId < hitsNum; hitId++)
    {
      const double hitX = hits[hitId].y;
      const double hitZ = hits[hitId].z;
      const double shift = (hitX - trackX0 - hitZ * trackDx);
      sum += exp(-shift * shift * sharpness);
    }
    values[trackId] = sum;
  }
}
*/