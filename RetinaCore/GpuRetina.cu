#include "hip/hip_runtime.h"
#include "GpuRetina.cuh"
#include <cstdio>

void getRetinaDx(
  const TrackProjection* tracks, 
  int tracksNum, 
  const Hit* hits,
  int hitsNum, 
  double sharpness,
  double *values
) {
  int trackId, hitId;
  for (trackId = 0; trackId < tracksNum; ++trackId)
  {
    if (trackId % 1000 == 0)
      printf("%d\n", trackId);
    const double trackX0 = tracks[trackId].x0;
    const double trackDx = tracks[trackId].dx;
    double sum = 0;
    for (hitId = 0; hitId < hitsNum; hitId++)
    {
      const double hitX = hits[hitId].x;
      const double hitZ = hits[hitId].z;
      const double shift = (hitX - trackX0 - hitZ * trackDx);
      sum += exp(-shift * shift * sharpness);
    }
    values[trackId] = sum;
  }
}
/*
void getRetinaDy(
  const TrackProjection* tracks, 
  int tracksNum, 
  const Hit* hits,
  int hitsNum, 
  double sharpness,
  double *values
) {
  int trackId, hitId;
  for (trackId = 0; trackId < tracksNum; ++trackId)
  {
    if (trackId % 1000 == 0)
      printf("%d\n", trackId);
    const double trackX0 = track[hitId].x;
    const double trackDx = track[hitId].dx;
    double sum = 0;
    for (hitId = 0; hitId < hitsNum; hitId++)
    {
      const double hitX = hits[hitId].y;
      const double hitZ = hits[hitId].z;
      const double shift = (hitX - trackX0 - hitZ * trackDx);
      sum += exp(-shift * shift * sharpness);
    }
    values[trackId] = sum;
  }
}
*/