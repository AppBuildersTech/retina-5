#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include "gpuRetina.cuh"
template<int BLOCK_SIZE>
__global__ void calculateResponces(
  TrackPure *grid, 
  Hit *hits, 
  int hits_size, 
  double sharpness, 
  double *responces
)
{
  __shared__ double sums[BLOCK_SIZE];
	int index = threadIdx.x;
  int tid = threadIdx.x;
  int blockId = blockIdx.x;
  double sum = 0;
  while (index < hits_size)
  {
    sum += calculateResponce(grid[blockId], hits[index], sharpness);
    index += BLOCK_SIZE;
  }
  sums[tid] = sum;
  __syncthreads();
  for (int s = BLOCK_SIZE >> 1; s != 0; s >>= 1)
  {
     if (tid < s)
       sums[tid] += sums[tid + s];
    __syncthreads();
  }
  if (tid == 0)
    responces[blockId] = sums[0];  
}

std::vector<double> gpuCalculateRetinaResponces(const std::vector<TrackPure>& grid,
  const std::vector<Hit>& hits,
  double sharpness
)
{
  std::vector<double> responces(grid.size());

  TrackPure* grid_gpu;
  hipMalloc( (void **) &grid_gpu, grid.size() * sizeof(TrackPure));
  hipMemcpy(grid_gpu, grid.data(), grid.size() * sizeof(TrackPure), hipMemcpyHostToDevice);
  
  Hit* hits_gpu;
  hipMalloc( (void **) &hits_gpu, hits.size() * sizeof(Hit));
  hipMemcpy(hits_gpu, hits.data(), hits.size() * sizeof(Hit), hipMemcpyHostToDevice);
  
  double *responces_gpu;
  hipMalloc( (void **) &responces_gpu, grid.size() * sizeof(double));
  
  calculateResponces<THREADS_PER_BLOCK> <<<grid.size(), THREADS_PER_BLOCK>>> (grid_gpu, hits_gpu, hits.size(), sharpness, responces_gpu);
  
  hipMemcpy(responces.data(), responces_gpu, grid.size() * sizeof(double), hipMemcpyDeviceToHost);
  
  hipFree(hits_gpu);
  hipFree(grid_gpu);
  hipFree(responces_gpu);
  
  return responces;
}
