#include "hip/hip_runtime.h"
#include "GpuRetina.cuh"
#include <cstdio>
template<int BLOCK_SIZE>
__global__ void calculateRetina2d(
  const TrackProjection* tracks, 
  int tracksNum, 
  const double* hitsX,
  const double* hitsZ,
  int hitsNum, 
  double sharpness,
  double *values
)
{
  int trackId = blockIdx.x;
  unsigned int tid = threadIdx.x;
  const double trackX0 = tracks[trackId].x0;
  const double trackDx = tracks[trackId].dx;
  double sum = 0;
  for (int hitId = tid; hitId < hitsNum; hitId += BLOCK_SIZE)
  {
    const double hitX = hitsX[hitId];
    const double hitZ = hitsZ[hitId];
    const double shift = (hitX - trackX0 - hitZ * trackDx);
    sum += exp(-shift * shift * sharpness);
  }
  __shared__ double sdata[BLOCK_SIZE];
  sdata[tid] = sum;
  for (unsigned int s = BLOCK_SIZE >> 1; s > 0; s >>= 1) 
  {
    __syncthreads();
    if (tid < s) 
    {
      sdata[tid] += sdata[tid + s];
    }
  }
  if (tid == 0)
  {
    values[trackId] = sdata[0];
  }
}

template<class T>
T* allocAndFetch(const T* data, int size)
{
  T* answer = nullptr;
  hipMalloc( (void**)&answer, sizeof(T) * size);
  hipMemcpy( answer, data, sizeof(T) * size, hipMemcpyHostToDevice);
  return answer;
}

void getRetina2dGpu(
  const TrackProjection* tracks, 
  int tracksNum, 
  const double* hitsX,
  const double* hitsZ,
  int hitsNum, 
  double sharpness,
  double *values
) {
  const int BLOCK_SIZE = 1 << 8;
  TrackProjection* tracksGpu = allocAndFetch(tracks, tracksNum);
  double* hitsXGpu = allocAndFetch(hitsX, hitsNum);
  double* hitsZGpu = allocAndFetch(hitsZ, hitsNum);
  double* valuesGpu = nullptr;
  hipMalloc( (void**)&valuesGpu, sizeof(double) * tracksNum);
  calculateRetina2d<BLOCK_SIZE><<<tracksNum, BLOCK_SIZE>>>(
    tracksGpu, 
    tracksNum, 
    hitsXGpu, 
    hitsZGpu, 
    hitsNum, 
    sharpness, 
    valuesGpu
  );
  hipMemcpy(values, valuesGpu, sizeof(double) * tracksNum, hipMemcpyDeviceToHost );
  hipFree(tracksGpu);
  hipFree(hitsXGpu);
  hipFree(hitsZGpu);
  hipFree(valuesGpu);
}

template<int BLOCK_SIZE>
__global__ void calculateRetina3d(
  const TrackPure* tracks, 
  int tracksNum, 
  const Hit* hits,
  int hitsNum, 
  double sharpness,
  double *values
)
{
  int trackId = blockIdx.x;
  unsigned int tid = threadIdx.x;
  const double trackX0 = tracks[trackId].x0;
  const double trackDx = tracks[trackId].dx;
  const double trackY0 = tracks[trackId].y0;
  const double trackDy = tracks[trackId].dy;
  double sum = 0;
  for (int hitId = tid; hitId < hitsNum; hitId += BLOCK_SIZE)
  {
    const float hitX = hits[hitId].x;
    const float hitY = hits[hitId].y;
    const float hitZ = hits[hitId].z;
    const double shiftX = (hitX - trackX0 - hitZ * trackDx);
    const double shiftY = (hitY - trackY0 - hitZ * trackDy);
    
    sum += exp(-(shiftX * shiftX + shiftY * shiftY) * sharpness);
  }
  __shared__ double sdata[BLOCK_SIZE];
  sdata[tid] = sum;
  __syncthreads();

  for (unsigned int s = BLOCK_SIZE >> 1; s > 0; s >>= 1) 
  {
    if (tid < s) 
    {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0)
  {
    values[trackId] = sdata[0];
  }
}

void getRetina3dGpu(
  const TrackPure* tracks, 
  int tracksNum, 
  const Hit* hitsX,
  int hitsNum, 
  double sharpness,
  double *values
) {
  const int BLOCK_SIZE = 1 << 8;
  TrackPure* tracksGpu = allocAndFetch(tracks, tracksNum);
  Hit* hitsGpu = allocAndFetch(hitsX, hitsNum);
  double* valuesGpu = nullptr;
  hipMalloc( (void**)&valuesGpu, sizeof(double) * tracksNum);
  calculateRetina3d<BLOCK_SIZE><<<tracksNum, BLOCK_SIZE>>>(
    tracksGpu, 
    tracksNum, 
    hitsGpu, 
    hitsNum, 
    sharpness, 
    valuesGpu
  );
  hipMemcpy( values, valuesGpu, sizeof(double) * tracksNum, hipMemcpyDeviceToHost );
  hipFree(tracksGpu);
  hipFree(hitsGpu);
  hipFree(valuesGpu);
}

