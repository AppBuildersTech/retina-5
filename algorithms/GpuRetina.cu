#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <algorithm>
#include <map>
#include <set>
#include <cstdlib>
#include <fstream> 

#include "Physics.h"
#include "HitsFinders.h"

#include "GpuRetina.cuh"

std::vector<double> getRetinaDx(const std::vector<TrackProjection>& tracks, const EventInfo& event, double sharpness);
{
  std::vector<double> values(tracks.size());
  size_t tracksNum = tracks.size();
  const std::vector<Hit>& hits = event.hits;
  size_t hitsNum = hits.size();
  
  for (size_t trackId = 0; trackId < tracksNum; ++trackId)
  {
    double sum = 0;
    for (size_t hitId = 0; hitId < hitsNum; hitId++)
    {
      sum += exp(-getDistanceDx(tracks[trackId], hits[hitId]) / sharpness);
    }
    values[trackId] = sum;
  }
  return values;
}


